#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>
#define N 2//64

__global__ void foo(int* p) {

  int* q;

  q = p;

  q[threadIdx.x] = 0;

}

int main() {
	int *c;
	int *dev_c;

	c = (int*)malloc(N*sizeof(int)); /* acessível apenas pela CPU função main e funções __host__ */

	for (int i = 0; i < N; ++i)
		c[i] = rand() %10+1;

	for (int i = 0; i < N; ++i)
		printf(" %d    :", c[i]);

	hipMalloc((void**)&dev_c, N*sizeof(int)); /* acessível apenas pela GPU funções __global__ */

	hipMemcpy(dev_c, c, N*sizeof(int), hipMemcpyHostToDevice);

	//foo<<<1, N>>>(dev_c);
	ESBMC_verify_kernel(foo,1,N,dev_c);

	hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

	printf ("\n");

	for (int i = 0; i < N; ++i){
		printf(" %d    :", c[i]);
		assert(c[i] != 0);
	}
	free(c);
	hipFree(dev_c);

	   return 0;
}
