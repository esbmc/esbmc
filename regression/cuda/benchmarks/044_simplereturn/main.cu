#include "hip/hip_runtime.h"
//pass
//--blockDim=64 --gridDim=64 --no-inline

#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <stdlib.h>
#include <assert.h>

#define N 2//64

__device__ int f(int x) {

  return x + 1;
}

__global__ void foo(int *y) {

	*y = f(2);

}

int main() {
	int *a = (int*)malloc(sizeof(int));
	int *dev_a;

	hipMalloc((void**)&dev_a, sizeof(int));
	
	//foo<<<1, N>>>(dev_a);
		ESBMC_verify_kernel(foo, 1, N, dev_a);

	hipMemcpy(a, dev_a, sizeof(int), hipMemcpyDeviceToHost);

//	printf("%d", *a);

	assert(*a==3);

	free(a);
	hipFree(dev_a);

	return 0;
}
