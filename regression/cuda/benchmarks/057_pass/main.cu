#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <stdlib.h>
#include <assert.h>

#define N 2//64

__device__ int f(int x) {
	
	return x + 2;
}

__global__ void foo(int *y, int x) {

	*y = f(x);

}

int main() {
	int a=2;
	int b=0;
	int *dev_a;

	hipMalloc((void**)&dev_a, sizeof(int));

	hipMemcpy(dev_a, &a, sizeof(int), hipMemcpyHostToDevice);

	//foo<<<1, N>>>(dev_a, a);
		ESBMC_verify_kernel_intt(foo, 1, N, dev_a, a);

	hipMemcpy(&b, dev_a, sizeof(int), hipMemcpyDeviceToHost);

	assert (b == a+2); 

	hipFree(dev_a);

	return 0;
}
