#include "hip/hip_runtime.h"
//fail
//--blockDim=64 --gridDim=64 --no-inline

#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <assert.h>

#define N 2//64

__device__ void bar(float x) {
	assert(0);
}

__global__ void foo(int* A) {

  bar(A[0]);

}

int main(){

	int *b;
	int *dev_b;

	b = (int*)malloc(N*sizeof(int));

	for (int i = 0; i < N; ++i){
		b[i] = i+1;
		printf(" %d; ", b[i]);
	}

	hipMalloc((void**)&dev_b, N*sizeof(float));

	hipMemcpy(dev_b, b, N*sizeof(float), hipMemcpyHostToDevice);

		//foo<<<1,N>>>(dev_b);
		ESBMC_verify_kernel(foo,1,N,dev_b);	

	free(b);
	hipFree(dev_b);
}
