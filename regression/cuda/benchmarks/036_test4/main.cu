#include "hip/hip_runtime.h"
//pass: checka a função device (comparar com o cuda69_test2)
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>
#define N 2//64

__device__ void bar(int* p) {
  p[threadIdx.x] = 0;
  //printf(" %d; ", p[threadIdx.x]);
}

__global__ void foo(int* p) {

  bar(p);

}

int main() {
	int *c;
	int *dev_c;
	c = (int*)malloc(N*sizeof(int));

	for (int i = 0; i < N; ++i)
		c[i] = rand() %10+1;

	for (int i = 0; i < N; ++i)
		printf(" %d; ", c[i]);

	hipMalloc((void**)&dev_c, N*sizeof(int));

	hipMemcpy(dev_c, c, N*sizeof(int), hipMemcpyHostToDevice);

	//foo<<<1, N>>>(dev_c);
	ESBMC_verify_kernel(foo,1,N,dev_c);

	hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

	printf ("\n");

	for (int i = 0; i < N; ++i){
		printf(" %d; ", c[i]);
		assert(c[i]==0);
	}

	free(c);
	hipFree(dev_c);

	   return 0;
}
