#include "hip/hip_runtime.h"
//fail
//--blockDim=64 --gridDim=64 --no-inline
#include <stdio.h>
#include "hip/hip_runtime_api.h"
#define N 1

__device__ void bar (int *p){

    int a = 0;

    p = &a;
}

__global__ void foo (int* p, int* q){

    if (*p > 10){
        bar(p);
        //*p = 23; // remove this comment to see that the __device__ function does not work
    }
    else {
        bar(q);
        //*q = 23; // remove this comment to see that the __device__ function does not work
    }
}

int main(){
	int *a, *b;
	int *dev_a, *dev_b;
	int size = N*sizeof(int);

	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);

	a = (int*)malloc(size);
	b = (int*)malloc(size);

	for (int i = 0; i < N; i++)
		a[i] = 1;

	for (int i = 0; i < N; i++)
		b[i] = 1;

	hipMemcpy(dev_a,a,size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,size, hipMemcpyHostToDevice);

	printf("a:  ");
	for (int i = 0; i < N; i++)
		printf("%d	", a[i]);

	printf("\nb:  ");
	for (int i = 0; i < N; i++)
		printf("%d	", b[i]);

	// foo<<<1,2>>>(dev_a,dev_b);
	ESBMC_verify_kernel(foo,1,2,dev_a,dev_b);

	hipMemcpy(a,dev_a,size,hipMemcpyDeviceToHost);
	hipMemcpy(b,dev_b,size,hipMemcpyDeviceToHost);

	printf("\nnew a and new b:\n");

	printf("a:  ");
	for (int i = 0; i < N; i++)
		printf("%d	", a[i]);

	printf("\nb:  ");
	for (int i = 0; i < N; i++)
		printf("%d	", b[i]);

	assert(b[0]!=1);

	free(a); free(b);

	hipFree(dev_a);
	hipFree(dev_b);

	return 0;
}
