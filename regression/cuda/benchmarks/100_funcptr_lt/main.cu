#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

#define N 2

typedef float(*funcType)(float*, unsigned int);

__device__ float multiplyByTwo(float *v, unsigned int tid)
{
    return v[tid] * 2.0f;
}

__device__ float divideByTwo(float *v, unsigned int tid)
{
    return v[tid] * 0.5f;
}

// Static pointers to device functions

	__device__ funcType p_mul_func = multiplyByTwo;

	__device__ funcType p_div_func = divideByTwo;

__global__ void foog(float *v, funcType f, funcType g, unsigned int size)
{
	assert(f == divideByTwo);
	assert(g == multiplyByTwo);

    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    funcType h;
    
    if (f >= g)
      h = f;
    else
      h = g;

    if (tid < size)
    {
        v[tid] = (*h)(v, tid);
    }
}

int main(){
	float* w;
	float* dev_w;

	int size = N*sizeof(float);

	w =(float*) malloc(size);

	for (int i = 0; i < N; ++i){
		w[i] = i;
	}

	hipMalloc((void**)&dev_w, size);

	hipMemcpy(dev_w,w, size,hipMemcpyHostToDevice);

	funcType host_f;

	hipMemcpyFromSymbol( &host_f, HIP_SYMBOL(&p_div_func), sizeof( funcType ), 0, hipMemcpyDeviceToHost);

	funcType dev_f = host_f;

	funcType host_g;

	hipMemcpyFromSymbol( &host_g, HIP_SYMBOL(&p_mul_func), sizeof( funcType ), 0, hipMemcpyDeviceToHost);

	funcType dev_g = host_g;

	// foog <<<1,N>>>(dev_w, dev_f, dev_g, N );

	hipDeviceSynchronize();

	hipMemcpy(w,dev_w,size,hipMemcpyDeviceToHost);

	printf("\nw:");
	for (int i = 0; i < N; ++i){
		printf(" %f	",	w[i]);
	}

	free(w);

	return 0;
}
