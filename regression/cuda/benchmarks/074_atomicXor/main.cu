#include "hip/hip_runtime.h"
//pass
//--blockDim=1024 --gridDim=1 --no-inline

#include <hip/hip_runtime_api.h>
#include <stdio.h>

#define N 2 //1024

__global__ void definitions (int* A, unsigned int* B, unsigned long long int* C)
{
  atomicXor(A,10);//1010 xor 0101 = 1111 /*xor looks for distinct bits*/

  atomicXor(B,7);//0111 xor 0101 = 0010

  atomicXor(C,5);//0101 xor 0101 = 0000
}

int main (){

	int a = 5;
	int *dev_a;

	hipMalloc ((void**) &dev_a, sizeof(int));

	hipMemcpy(dev_a, &a, sizeof(int),hipMemcpyHostToDevice);

	unsigned int b = 5;
	unsigned int *dev_b;

	hipMalloc ((void**) &dev_b, sizeof(unsigned int));

	hipMemcpy(dev_b, &b, sizeof(unsigned int),hipMemcpyHostToDevice);

	unsigned long long int c = 5;
	unsigned long long int *dev_c;

	hipMalloc ((void**) &dev_c, sizeof(unsigned long long int));

	hipMemcpy(dev_c, &c, sizeof(unsigned long long int),hipMemcpyHostToDevice);

	//definitions <<<1,N>>>(dev_a,dev_b,dev_c);
	ESBMC_verify_kernel(definitions,1,N,dev_a,dev_b,dev_c);

	hipMemcpy(&a,dev_a,sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(&b,dev_b,sizeof(unsigned int),hipMemcpyDeviceToHost);
	hipMemcpy(&c,dev_c,sizeof(unsigned long long int),hipMemcpyDeviceToHost);

	printf("A: %d\n", a);
	printf("B: %u\n", b);
	printf("C: %u\n", c);

	assert(a==5);
	assert(b==5);
	assert(c==5);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return 0;

}
