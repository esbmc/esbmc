
//xfail:BOOGIE_ERROR
//--blockDim=2 --gridDim=1 --no-inline
//Write by thread .+kernel.cu:8:4:
// to threadIdx.x != 0 we have 'data race'.


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

#define N 8 //2

__global__ void init_test(hiprandState *state, unsigned int *A) {
   hiprand_init(0, 0, 0, state);

   __syncthreads();

   A[threadIdx.x] =  hiprand(&state[threadIdx.x]);
//   if (threadIdx.x == 0) {
  //   A[0] = curand(state);
   //}
}

int main(){
	unsigned int *a;
	unsigned int *dev_a;
	hiprandState *dev_state; 

	int size = N*sizeof(unsigned int);

	a = (unsigned int*)malloc(size);
	hipMalloc ((void**) &dev_a, size);

	printf("old a:  ");
	for (int i = 0; i < N; i++)
		printf("%u	", a[i]);

	hipMalloc ( (void**) &dev_state, N*sizeof( hiprandState ) );

	// init_test<<<1,N>>>(dev_state, dev_a);

	hipMemcpy(a,dev_a,size,hipMemcpyDeviceToHost);

	printf("\nnew a:  ");
	for (int i = 0; i < N; i++) {
		printf("%u	", a[i]);
		//assert((a[i] == 0 || a[i] == 1)); // we can't put assert() here because we get random numbers
		// maybe we can check if they are > 0 or not NULL... ?
	}

	free(a);
	hipFree(&dev_a);
	hipFree(&dev_state);

	return 0;
}

