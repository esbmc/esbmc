#include "hip/hip_runtime.h"
//pass
//--blockDim=1024 --gridDim=1 --no-inline

#include <hip/hip_runtime_api.h>
#include <stdio.h>

#define N 2 //1024

__global__ void definitions (int* A, unsigned int* B, unsigned long long int* C)
{
  atomicCAS(A,2,11);

  atomicCAS(B,5,1);

  atomicCAS(C,7,3);
}

int main (){

	int a = 5;
	int *dev_a;

	hipMalloc ((void**) &dev_a, sizeof(int));

	hipMemcpy(dev_a, &a, sizeof(int),hipMemcpyHostToDevice);

	unsigned int b = 5;
	unsigned int *dev_b;

	hipMalloc ((void**) &dev_b, sizeof(unsigned int));

	hipMemcpy(dev_b, &b, sizeof(unsigned int),hipMemcpyHostToDevice);

	unsigned long long int c = 5;
	unsigned long long int *dev_c;

	hipMalloc ((void**) &dev_c, sizeof(unsigned long long int));

	hipMemcpy(dev_c, &c, sizeof(unsigned long long int),hipMemcpyHostToDevice);

//	definitions <<<1,N>>>(dev_a,dev_b,dev_c);
	ESBMC_verify_kernel(definitions,1,N,dev_a,dev_b,dev_c);

	hipMemcpy(&a,dev_a,sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(&b,dev_b,sizeof(unsigned int),hipMemcpyDeviceToHost);
	hipMemcpy(&c,dev_c,sizeof(unsigned long long int),hipMemcpyDeviceToHost);

	assert(a==5);
	assert(b==1);	
	assert(c==5);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return 0;

}
