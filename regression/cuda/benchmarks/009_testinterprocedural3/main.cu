#include "hip/hip_runtime.h"
//fail
//--blockDim=64 --gridDim=64 --no-inline

#include <stdio.h>
#include <hip/hip_runtime_api.h>
#define N 1

__device__ void baz (int p []){
    int a;

    p = &a;
}

__device__ void bar (int *p){

    int a = 2;

    p = &a;
}


__global__ void foo (int* p, int* q){

    __shared__ int sharedArr  [100];

    __shared__ int sharedArr2 [50];

    bar(p);

    baz (sharedArr);

    bar(q);

    if (*q){
        baz(sharedArr2);
    }

    //*p = 23; *q = 23; // remove this comment to see that the __device__ function does not work
}

int main(){
	int *a, *b;
	int *dev_a, *dev_b;
	int size = N*sizeof(int);

	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);

	a = (int*)malloc(size);
	b = (int*)malloc(size);

	for (int i = 0; i < N; i++)
		a[i] = 1;

	for (int i = 0; i < N; i++)
		b[i] = 1;

	hipMemcpy(dev_a,a,size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,size, hipMemcpyHostToDevice);

	printf("a:  ");
	for (int i = 0; i < N; i++)
		printf("%d	", a[i]);

	printf("\nb:  ");
	for (int i = 0; i < N; i++)
		printf("%d	", b[i]);

	// foo<<<1,2>>>(dev_a,dev_b);
	ESBMC_verify_kernel(foo,1,2,dev_a,dev_b);
	
	hipMemcpy(a,dev_a,size,hipMemcpyDeviceToHost);
	hipMemcpy(b,dev_b,size,hipMemcpyDeviceToHost);

	printf("\nnew a and new b:\n");

	printf("a:  ");
	for (int i = 0; i < N; i++)
		printf("%d	", a[i]);

	printf("\nb:  ");
	for (int i = 0; i < N; i++)
		printf("%d	", b[i]);

	assert(a[0]!=1);
	assert(b[0]!=1);

	free(a); free(b);

	hipFree(dev_a);
	hipFree(dev_b);

	return 0;
}
