//pass
//--blockDim=512 --gridDim=1 --no-inline


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <assert.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define N 2 //512

__global__ void curand_test(hiprandState *state, float *A) {
   A[threadIdx.x] =  hiprand(&state[threadIdx.x]); // the pseudo random number returned by 'curand' is an unsigned int
}

int main() {
	float *a;
	float *dev_a;
	hiprandState *dev_state; // is not necessary to initialize dev_state because it is a receptor in the function.

	int size = N*sizeof(float);

	a = (float*)malloc(size);
	hipMalloc ((void**) &dev_a, size);

	printf("old a:  ");
	for (int i = 0; i < N; i++)
		printf("%f	", a[i]);

	hipMalloc ( (void**) &dev_state, N*sizeof( hiprandState ) );

	// curand_test<<<1,N>>>(dev_state, dev_a);
    // ESBMC_verify_kernel(curand_test,1,N,dev_state,dev_a);
	
	hipMemcpy(a,dev_a,size,hipMemcpyDeviceToHost);

	printf("\nnew a:  ");
	for (int i = 0; i < N; i++) {
		printf("%f	", a[i]);
		//assert((a[i] == 0 || a[i] == 1)); // we can't put assert() here because we get random numbers
		// maybe we can check if they are > 0 or not NULL... ?
	}
	
	free(a);
	hipFree(dev_a);
	hipFree(dev_state);

	return 0;
}
