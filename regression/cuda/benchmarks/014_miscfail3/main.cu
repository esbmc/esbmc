#include "hip/hip_runtime.h"
//xfail:BOOGIE_ERROR
//main.cu: error: possible read-write race
//however, this didn't happen in the tests
//altough in CUDA providing the inline keyword should still keep a copy of the function around,
//this kind of access is considered a error by ESBMC
//ps: the values from A[N-1-offset) to A[N-1] always will receive unpredictable values,
//because they acess values because they access memory positions that were not initiated

#include <stdio.h>
#include <hip/hip_runtime_api.h>
#define tid threadIdx.x
#define N 2//1024

__device__ inline void inlined(int *A, int offset)
{
   int temp = A[tid + offset];
   A[tid] += temp;
}

__global__ void inline_test(int *A, int offset) {
  inlined(A, offset);
}

int main(){

	int *a;
	int *dev_a;
	int size = N*sizeof(int);

	hipMalloc((void**)&dev_a, size);

	a = (int*)malloc(N*size);

	for (int i = 0; i < N; i++)
		a[i] = i;

	hipMemcpy(dev_a,a,size, hipMemcpyHostToDevice);

	printf("a:  ");

	//inline_test<<<1,N>>>(dev_a, 2);		//you can change this offset for tests
	ESBMC_verify_kernel_intt(inline_test, 1, N, dev_a, 1);
	
	hipMemcpy(a,dev_a,size,hipMemcpyDeviceToHost);

	printf("\nFunction Results:\n   ");

	free(a);

	hipFree(dev_a);

	return 0;
}
