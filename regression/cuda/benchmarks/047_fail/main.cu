#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime_api.h"
#include <assert.h>

#define N 2//64

__global__ void foo(int *c) {
  int b, a;
  a = 2;
  b = 3;
  c[threadIdx.x]= a+b;
  __syncthreads ();
}

int main(){
	int *a;
	int *dev_a;

	a = (int*)malloc(N*sizeof(int));

	hipMalloc((void**)&dev_a, N*sizeof(int));

	hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);

	//foo<<<1, N>>>(dev_a);
	ESBMC_verify_kernel(foo,1,N,dev_a);

	hipMemcpy(a, dev_a, N*sizeof(int), hipMemcpyDeviceToHost);

	for (int t=0;t<N;t++){
		//printf ("%d ", a[t]);
		assert(a[t]!=5);
	}

	hipFree(dev_a);
	free(a);
	return 0;
}
