#include "hip/hip_runtime.h"
//pass
//--blockDim=1024 --gridDim=1 --no-inline

#include <hip/hip_runtime_api.h>
#include <stdio.h>

#define N 2 //1024

__global__ void definitions (int* A, unsigned int* B, unsigned long long int* C, float* D)
{
	atomicAdd(A,10);

	atomicAdd(B,10);

	atomicAdd(C,10);

	atomicAdd(D,10);

}

int main (){

	int a = 5;
	int *dev_a;

	hipMalloc ((void**) &dev_a, sizeof(int));

	hipMemcpy(dev_a, &a, sizeof(int),hipMemcpyHostToDevice);

	unsigned int b = -5;
	unsigned int *dev_b;

	hipMalloc ((void**) &dev_b, sizeof(unsigned int));

	hipMemcpy(dev_b, &b, sizeof(unsigned int),hipMemcpyHostToDevice);

	unsigned long long int c = 0;
	unsigned long long int *dev_c;

	hipMalloc ((void**) &dev_c, sizeof(unsigned long long int));

	hipMemcpy(dev_c, &c, sizeof(unsigned long long int),hipMemcpyHostToDevice);

	float d = 10;
	float *dev_d;

	hipMalloc ((void**) &dev_d, sizeof(float));

	hipMemcpy(dev_d, &d, sizeof(float),hipMemcpyHostToDevice);

//	definitions <<<1,N>>>(dev_a,dev_b,dev_c,dev_d);
	ESBMC_verify_kernel_four(definitions,1,N,dev_a,dev_b,dev_c,dev_d);

	hipMemcpy(&a,dev_a,sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(&b,dev_b,sizeof(unsigned int),hipMemcpyDeviceToHost);
	hipMemcpy(&c,dev_c,sizeof(unsigned long long int),hipMemcpyDeviceToHost);
	hipMemcpy(&d,dev_d,sizeof(float),hipMemcpyDeviceToHost);

	assert(a==25);
	assert(b==15);
	assert(c==20);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipFree(dev_d);
	return 0;

}
