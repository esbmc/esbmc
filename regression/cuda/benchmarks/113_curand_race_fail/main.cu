
//xfail:BOOGIE_ERROR
//--blockDim=2 --gridDim=1 --no-inline
//Write by thread .+kernel\.cu:8:21:


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <assert.h>

#define N 4

__global__ void curand_test(hiprandState *state, float *A) { // test: replace curandState for curandStateXORWOW_t
   A[threadIdx.x] = hiprand_uniform(state);
}

int main(){
	float *a;
	float *dev_a;
	hiprandState *dev_state; 
	
	int size = N*sizeof(float);

	a = (float*)malloc(size);
	hipMalloc ((void**) &dev_a, size);

	printf("old a:  ");
	for (int i = 0; i < N; i++)
		printf("%f	", a[i]);

	hipMalloc ( (void**) &dev_state, N*sizeof( hiprandState ) );

	//curand_test<<<1,N>>>(dev_state, dev_a);

	hipMemcpy(a,dev_a,size,hipMemcpyDeviceToHost);

	printf("\nnew a:  ");
	for (int i = 0; i < N; i++) {
		printf("%f	", a[i]);
		assert(a[i] == a[i+1]);
	}

	free(a);
	hipFree(&dev_a);
	hipFree(&dev_state);

	return 0;
}
