#include "hip/hip_runtime.h"
//xfail:BUGLE_ERROR
//--gridDim=1 --blockDim=32 --no-inline

//This kernel is racy: memset is called with variable length.
//#define memset(dst,val,len) __builtin_memset(dst,val,len)

#define N 2//32

#include <stdio.h>
#include <hip/hip_runtime_api.h>

__device__ int bar(void){
	int value;
	return value;
}

__global__ void kernel(uint4 *out) {
  uint4 vector;
  int len = bar();
   memset(&vector, 5, len); /*modify manually the value of len to see the bugs*/
  out[threadIdx.x] = vector;
}

int main(){
	uint4 *a;
	uint4 *dev_a;
	int size = N*sizeof(uint4);

	a = (uint4*)malloc(size);

	/* initialization of a */
	for (int i = 0; i < N; i++) {
		a[i].x = i; a[i].y = i; a[i].z = i, a[i].w = i;
	}

	hipMalloc((void**)&dev_a, size);

	hipMemcpy(dev_a,a,size, hipMemcpyHostToDevice);

	// kernel<<<1,N>>>(dev_a);
	ESBMC_verify_kernel(kernel,1,N,dev_a);

	hipMemcpy(a,dev_a,size,hipMemcpyDeviceToHost);

	hipFree(dev_a);
	free(a);
	return 0;
}
