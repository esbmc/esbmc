#include "hip/hip_runtime.h"
//assertion
//--blockDim=64 --gridDim=64 --no-inline

#include "hip/hip_runtime_api.h"
#include <stdio.h>

#define N 1

__device__ void bar (int *p){

	int a =2;
	
	p = &a;
	assert(*p == 2);
}

__global__ void foo (int* p, int* q){

    bar(p);

    bar(q);
	assert(*p == 2);
    //*p = 23; *q = 23; // remove this comment to see that the __device__ function does not work
}

int main(){
	int *a, *b;
	int *dev_a, *dev_b;
	int size = N*sizeof(int);

	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);

	a = (int*)malloc(size);
	b = (int*)malloc(size);

	for (int i = 0; i < N; i++)
		a[i] = 1;

	for (int i = 0; i < N; i++)
		b[i] = 1;

	hipMemcpy(dev_a,a,size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,size, hipMemcpyHostToDevice);

	// foo<<<1,2>>>(dev_a,dev_b);
	ESBMC_verify_kernel(foo,1,2,dev_a,dev_b);

	hipMemcpy(a,dev_a,size,hipMemcpyDeviceToHost);
	hipMemcpy(b,dev_b,size,hipMemcpyDeviceToHost);

	free(a); free(b);

	hipFree(dev_a);
	hipFree(dev_b);

	return 0;
}
