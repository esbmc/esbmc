#include "hip/hip_runtime.h"
//pass: checka se o parâmetro é passado com sucesso
//--blockDim=1024 --gridDim=1 --no-inline
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

#define N 2

__device__ float multiplyByTwo(float *v, unsigned int tid) {

    return v[tid] * 2.0f;
}

__device__ float divideByTwo(float *v, unsigned int tid) {

    return v[tid] * 0.5f;
}

typedef float(*funcType)(float*, unsigned int);

__global__ void foo(float *v, funcType* f, unsigned int size)
{

	//*** __requires(f == multiplyByTwo | f == divideByTwo); ****/
	/************************************************************/
	assert(*f == divideByTwo || *f == multiplyByTwo);
	/************************************************************/

	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        v[tid] = (*f)(v, tid);
    }
}

int main (){

	float* w;
	float* dev_w;

	int size = N*sizeof(float);

	w =(float*) malloc(size);

	for (int i = 0; i < N; ++i){
		w[i] = i;
	}

	hipMalloc((void**)&dev_w, size);

	hipMemcpy(dev_w,w, size,hipMemcpyHostToDevice);

	funcType* g;
	funcType* dev_g;
	g =(funcType*) malloc(sizeof(funcType));

	//*g = multiplyByTwo;
	*g = divideByTwo;

	hipMalloc((void**)&dev_g, sizeof(funcType));

	hipMemcpy(dev_g, g, sizeof(funcType),hipMemcpyHostToDevice);

	// foo <<<1,N>>>(dev_w, dev_g, N );

	hipMemcpy(w,dev_w,size,hipMemcpyDeviceToHost);

	hipMemcpy(g,dev_g,sizeof(funcType),hipMemcpyDeviceToHost);

	printf("\nw:");
	for (int i = 0; i < N; ++i){
		printf(" %f	",	w[i]);
	}

	//printf ("\n (float) functype: %f", divideByTwo);
	free(w);
	hipFree(dev_w);
	hipFree(dev_g);

	return 0;
}
