#include "hip/hip_runtime.h"
//fail
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>
#define N 2//64

__global__ void foo(int* p) {

  p[threadIdx.x] = 0;

}

int main() {
	int *c;
	int *dev_c;
	c = (int*)malloc(N*sizeof(int));

	for (int i = 0; i < N; ++i)
		c[i] = rand() %5+1;

	/*for (int i = 0; i < N; ++i)
		printf(" %d  ", c[i]);
	*/
	hipMalloc((void**)&dev_c, N*sizeof(int));

	hipMemcpy(dev_c, c, N*sizeof(int), hipMemcpyHostToDevice);

	//foo<<<1, N>>>(dev_c);
	ESBMC_verify_kernel(foo,1,N,dev_c);

	hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

	//printf ("\n");

	for (int i = 0; i < N; ++i){
	//		printf(" %d  ", c[i]);
			assert(c[i] != 0);
	}
	free(c);
	hipFree(dev_c);

	   return 0;
}
