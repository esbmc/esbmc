#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>

__global__ void Asum(int *a, int *b, int *c){
	*c = *a + *b;
}

int main(void){
	int a, b, c;
	int *dev_a, *dev_b, *dev_c;		//These are pointers to a memory slot ON DEVICE
	int size = sizeof(int);			//memory size in bytes

	hipMalloc((void**)&dev_a,size);	//hipMalloc() allocates a memory slot on device (GPU memory)
						//this slot equals size bytes
						//void** assures that pointers won't have trouble getting a variable that is not an int type
						//dev_a now points to the allocated slot
	hipMalloc((void**)&dev_b,size);
	hipMalloc((void**)&dev_c,size);	//conclusion: pointers are referencing a position that is avaliable from DEVICE
						//a, b and c positions are not avaliable from device, a priori
	a = 2;
	b = 7;
	c = 8;

	hipMemcpy(dev_a,&a,size, hipMemcpyHostToDevice);	//note that &a is used
	hipMemcpy(dev_b,&b,size, hipMemcpyHostToDevice);	//hipMemcpy(*destiny, *source, size, hipMemcpyKind)

	//	Asum<<<1,1>>>(dev_a,dev_b,dev_c);
	ESBMC_verify_kernel(Asum, 1,2,dev_a,dev_b, dev_c);

	hipMemcpy(&c,dev_c,size,hipMemcpyDeviceToHost);

	printf("a + b = %d\n", c);

	assert(c != a+b);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}
