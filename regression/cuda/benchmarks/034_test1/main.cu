#include "hip/hip_runtime.h"
//pass
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>

#define N 2//64

__global__ void foo(int* p) {

  p[threadIdx.x] = 0;

}

int main() {
	int *c;
	int *dev_c;
	c = (int*)malloc(N*sizeof(int));

	for (int i = 0; i < N; ++i)
		c[i] = rand() %10+1;

	hipMalloc((void**)&dev_c, N*sizeof(int));

	hipMemcpy(dev_c, c, N*sizeof(int), hipMemcpyHostToDevice);

		//foo<<<1, N>>>(dev_c);
		ESBMC_verify_kernel(foo,1,N,dev_c);

	hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < N; ++i)
			assert(c[i] == 0);
	
	free(c);
	hipFree(dev_c);

	   return 0;
}
