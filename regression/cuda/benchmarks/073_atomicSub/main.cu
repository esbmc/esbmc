#include "hip/hip_runtime.h"
//pass
//--blockDim=1024 --gridDim=1 --no-inline

#include <hip/hip_runtime_api.h>
#include <stdio.h>

#define N 2 //1024

__global__ void definitions (int* A, unsigned int* B)
{
	atomicSub(A,10);

	atomicSub(B,5);

}

int main (){

	int a = 5;
	int *dev_a;

	hipMalloc ((void**) &dev_a, sizeof(int));

	hipMemcpy(dev_a, &a, sizeof(int),hipMemcpyHostToDevice);

	unsigned int b = 10;
	unsigned int *dev_b;

	hipMalloc ((void**) &dev_b, sizeof(unsigned int));

	hipMemcpy(dev_b, &b, sizeof(unsigned int),hipMemcpyHostToDevice);

	//definitions <<<1,N>>>(dev_a,dev_b);
	ESBMC_verify_kernel(definitions,1,N,dev_a,dev_b);

	hipMemcpy(&a,dev_a,sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(&b,dev_b,sizeof(unsigned int),hipMemcpyDeviceToHost);

	printf("A: %d\n", a);
	printf("B: %u\n", b);

	assert(a==-15);
	assert(b==0);

	hipFree(dev_a);
	hipFree(dev_b);

	return 0;

}
