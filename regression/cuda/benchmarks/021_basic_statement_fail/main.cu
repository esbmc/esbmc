#include "hip/hip_runtime.h"
//fail: null-pointer
//--blockDim=1024 --gridDim=1 --no-inline
#include "hip/hip_runtime_api.h"
#include <stdio.h>
#include <assert.h>

#define tid (blockIdx.x * blockDim.x + threadIdx.x)
#define N 2//1024

__device__ void multiplyByTwo(float *v, unsigned int index)
{ 
    v[index] = v[index] * 2.0f;
}

__device__ void divideByTwo(float *v, unsigned int index)
{
    v[index] = v[index] * 0.5f;
}

typedef void(*funcType)(float*, unsigned int);

__global__ void foo(float *v, unsigned int i)
{
    funcType f;

    if (i == 1)
      f = multiplyByTwo;
    else if (i == 2)
      f = divideByTwo;
    else
      f = NULL;

    (*f)(v, tid);
}

int main(){

	unsigned int c; /*define se multiplicará ou dividirá por 2, deve ser 1 ou 2, para escolher a função*/
	float* v;
	float* a;
	float* dev_v;

		printf("Digite 1 para duplicar um vetor ou\nDigite 2 para dividir um vetor por 2: \n");
		scanf("%u", &c);

	v = (float*)malloc(N*sizeof(float));
	a = (float*)malloc(N*sizeof(float));

	for (int i = 0; i < N; ++i){
		v[i] = rand() %10+1;
		printf(" %.1f; ", v[i]);
	}

	printf("\n");

	hipMalloc((void**)&dev_v, N*sizeof(float));

	hipMemcpy(dev_v, v, N*sizeof(float), hipMemcpyHostToDevice);

	//foo<<<1, N>>>(dev_v, c);
	ESBMC_verify_kernel_fuintt(foo, 1, N, dev_v, c);

	hipMemcpy(a, dev_v, N*sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 0; i < N; ++i){
		printf(" %.1f; ", a[i]);
		if (c==1)
			assert(a[i]==2*v[i]);
		else if (c==2)
			assert(a[i]==0.5*v[i]);
	}

	free(v); free(a);
	hipFree(dev_v);

	return 0;
}
