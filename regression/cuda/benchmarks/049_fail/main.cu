#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include "hip/hip_runtime_api.h"

#define N 2

__global__ void foo(int* p) {
    p[threadIdx.x] = 2;
    __syncthreads();
}

int main(){

	int *a;
	int *dev_a;
	int size = N*sizeof(int);

	hipMalloc((void**)&dev_a, size);

	a = (int*)malloc(N*size);

	hipMemcpy(dev_a,a,size, hipMemcpyHostToDevice);

	//foo<<<1,N>>>(dev_a);
	ESBMC_verify_kernel(foo,1,N,dev_a);

	hipMemcpy(a,dev_a,size,hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++){
		assert(a[i]==1);
	}

	free(a);

	hipFree(dev_a);

	return 0;
}
