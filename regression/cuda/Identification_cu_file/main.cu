#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
using namespace std;

__global__ void kernel(void) {
  printf("hello world gpu \n");
}
int main() {
  kernel<<<1, 1>>>();
    hipError_t cudaStatus;

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    }

  return 0;
}
