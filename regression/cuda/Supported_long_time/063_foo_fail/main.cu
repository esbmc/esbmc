#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime_api.h"
#include <assert.h>
#define N 2 //16

__device__ int bar(int x) {

	return x + 1;
}

__global__ void foo(int *A) {

	A[threadIdx.x] = bar(threadIdx.x);
}


int main () {

	int *a;
	int *dev_a;
	int size = N*sizeof(int);

	hipMalloc((void**)&dev_a, size);

	a = (int*)malloc(size);

	for (int i = 0; i < N; i++)
		a[i] = 1;

//	foo<<<1,N>>>(dev_a);
	ESBMC_verify_kernel(foo, 1, N, dev_a);

	hipMemcpy(a,dev_a,size,hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++){
		assert(!(a[i]== (i+1)));
	}

	free(a);
	hipFree(dev_a);

	return 0;
}
