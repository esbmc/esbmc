#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime_api.h>
#include <assert.h>

#define N 2//(64*64)//(2048*2048)
#define THREADS_PER_BLOCK 2//512

__global__ void Asum(int *a, int *b, int *c){
	int index = threadIdx.x;
	c[index] = a[index] + b[index];
}

int main(void){
	int *a, *b, *c;
	int *dev_a, *dev_b, *dev_c;
	int size = N*sizeof(int);

	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c,size);

	a = (int*)malloc(size);
	b = (int*)malloc(size);
	c = (int*)malloc(size);

	for (int i = 0; i < N; i++)
		a[i] = 10;

	for (int i = 0; i < N; i++)
		b[i] = 10;

	hipMemcpy(dev_a,a,size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,size, hipMemcpyHostToDevice);

	printf("a:  ");
	for (int i = 0; i < N; i++)
		printf("%d	", a[i]);

	printf("\nb:  ");
	for (int i = 0; i < N; i++)
		printf("%d	", b[i]);

	//Asum<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(dev_a,dev_b,dev_c);
	ESBMC_verify_kernel(Asum, N/THREADS_PER_BLOCK,THREADS_PER_BLOCK,dev_a,dev_b,dev_c);

	hipMemcpy(c,dev_c,size,hipMemcpyDeviceToHost);

	printf("\nResultado da soma de a e b eh:\n   ");

	for (int i = 0; i < N; i++){
		printf("%d	", c[i]);
		assert(c[i]!=a[i]+b[i]);
	}

	free(a); free(b); free(c);

	hipFree(dev_a);
	hipFree(dev_c);
	hipFree(dev_b);

	return 0;

}
