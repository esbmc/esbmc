#include "hip/hip_runtime.h"
//fail: assertion
//--blockDim=1024 --gridDim=1 --no-inline

#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <assert.h>
#define N 2//1024

__global__ void foo(float *A, int sz) {
	assert(sz == blockDim.x);
	for(int i = threadIdx.x; i < 1*sz; i += sz) {
		A[i] *= 2.0f;
  }
}

int main() {

    float *a;
    float *b;
    float *dev_a;
    int size = N*sizeof(float);

    hipMalloc((void**)&dev_a, size);

    a = (float*)malloc(N*size);
    b = (float*)malloc(N*size);

    for (int i = 0; i < N; i++)
        a[i] = i;

    hipMemcpy(dev_a,a,size, hipMemcpyHostToDevice);

	     	//   foo<<<1,N>>>(dev_a, N);
		ESBMC_verify_kernel_fuintint(foo, 1, N, dev_a, N);

    hipMemcpy(b,dev_a,size,hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++){
        assert(b[i]!=2*a[i]);
    }

    free(a); free(b);
    hipFree(dev_a);

    return 0;
}
