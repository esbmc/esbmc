#include "hip/hip_runtime.h"
//pass
//--blockDim=512 --gridDim=1 --warp-sync=32 --no-inline

#include <hip/hip_runtime_api.h>
#include <stdio.h>

#define N 2


__global__ void scan (int* A)
{
	int tid = threadIdx.x;
	unsigned int lane = tid & 31;

	if (lane >= 1) A[tid] = A[tid - 1] + A[tid];
	if (lane >= 2) A[tid] = A[tid - 2] + A[tid];
	if (lane >= 4) A[tid] = A[tid - 4] + A[tid];
	if (lane >= 8) A[tid] = A[tid - 8] + A[tid];
	if (lane >= 16) A[tid] = A[tid - 16] + A[tid];
}

int main(){
	int *a;
	int *dev_a;
	int size = N*sizeof(int);

	hipMalloc((void**)&dev_a, size);

	a = (int*)malloc(size);

	for (int i = 0; i < N; i++)
		a[i] = i;

	hipMemcpy(dev_a,a,size, hipMemcpyHostToDevice);

	printf("old a:  ");
	for (int i = 0; i < N; i++)
		printf("%d	", a[i]);

	//scan<<<1,N>>>(dev_a);
	ESBMC_verify_kernel(scan, 1, 2, dev_a);

	hipMemcpy(a,dev_a,size,hipMemcpyDeviceToHost);

	printf("\nnew a:  ");
	for (int i = 0; i < N; i++)
		printf("%d	", a[i]);

	free(a);

	hipFree(dev_a);


	return 0;
}
