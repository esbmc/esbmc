#include "hip/hip_runtime.h"
//pass
//--blockDim=2 --gridDim=2
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>

#define N 2

struct S {
  int * p;
};

__global__ void foo(int * A) {

  S myS;
  myS.p = A;
  int * q;
  q = myS.p;
  q[threadIdx.x + blockDim.x*blockIdx.x] = threadIdx.x;

}

int main() {
	int *a;
	int *dev_a;
	int size = N*sizeof(int);

	hipMalloc((void**)&dev_a, size);

	a = (int*)malloc(size);

	for (int i = 0; i < N; i++)
		a[i] = 5;

	hipMemcpy(dev_a,a,size, hipMemcpyHostToDevice);

	//foo<<<1,N>>>(dev_a);
	ESBMC_verify_kernel(foo,1,N,dev_a);

	hipMemcpy(a,dev_a,size,hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++) {
		assert(a[i] == 0 || a[i] == 1);
	}

	free(a);

	hipFree(dev_a);

	return 0;
}
