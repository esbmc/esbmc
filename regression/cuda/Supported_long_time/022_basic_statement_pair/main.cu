#include "hip/hip_runtime.h"
//pass: checka a declaração de uma variável do tipo "ponteiro pra função" dentro do kernel
//--blockDim=1024 --gridDim=1 --no-inline
#include "hip/hip_runtime_api.h"
#include <stdio.h>
#include <assert.h>

#define tid (blockIdx.x * blockDim.x + threadIdx.x)
#define N 2//1024

__device__ void multiplyByTwo(float *v, unsigned int index)
{  
	v[index] = v[index] * 2.0f;
}

__device__ void divideByTwo(float *v, unsigned int index)
{  
    v[index] = v[index] * 0.5f;
}

typedef void(*funcType)(float*, unsigned int);

__global__ void foo(float *v, unsigned int i)
{
    	assert(i == 1 | i == 2);
    	
    funcType f;

    if (i == 1)
      f = multiplyByTwo;
    else if (i == 2)
      f = divideByTwo;
    else
      f = NULL;

    (*f)(v, tid);
}

int main(){

	unsigned int c = 1; /*define se multiplicará ou dividirá por 2, deve ser 1 ou 2, para escolher a função*/
	float* v;
	float* a;
	float* dev_v;

	v = (float*)malloc(N*sizeof(float));
	a = (float*)malloc(N*sizeof(float));

	for (int i = 0; i < N; ++i){
		v[i] = rand() %10+1;
		printf(" %.1f; ", v[i]);
	}

	printf("\n");

	hipMalloc((void**)&dev_v, N*sizeof(float));

	hipMemcpy(dev_v, v, N*sizeof(float), hipMemcpyHostToDevice);

		//foo<<<1, N>>>(dev_v, c);
		ESBMC_verify_kernel_fuintt(foo, 1, N, dev_v, c);

	hipMemcpy(a, dev_v, N*sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 0; i < N; ++i){
		printf(" %.1f; ", a[i]);
		if (c==1)
			assert(a[i]==2*v[i]);
		else if (c==2)
			assert(a[i]==0.5*v[i]);
	}

	free(v); free(a);
	hipFree(dev_v);

	return 0;
}
