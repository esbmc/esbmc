#include "hip/hip_runtime.h"
//pass
//--blockDim=64 --gridDim=64 --no-inline

#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <assert.h>

#define DIM 2 //64
#define N 2// DIM*DIM

typedef struct myfloat4{
  float x,y,z,w;
} myfloat4;

__global__ void k(float * i0) {
  myfloat4 f4;
  f4.x = 2;
  i0[threadIdx.x + blockDim.x*blockIdx.x] = f4.x;
}

int main(){
	float *a;
	float *dev_a;
	int size = N*sizeof(float);

	hipMalloc((void**)&dev_a, size);

	a = (float*)malloc(size);

	for (int i = 0; i < N; i++)
		a[i] = 5;

	hipMemcpy(dev_a,a,size, hipMemcpyHostToDevice);

		//k<<<DIM,DIM>>>(dev_a);
		ESBMC_verify_kernel_f(k,1,N,dev_a);

	hipMemcpy(a,dev_a,size,hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++) {
		assert(a[i] == 2);
	}

	free(a);

	hipFree(dev_a);

	return 0;
}
