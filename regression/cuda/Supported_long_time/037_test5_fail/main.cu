#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>
#define N 2//64

__device__ int* bar(int* p) {
  return p;
}

__global__ void foo(int* p) {

  int* q = bar(p);

  q[threadIdx.x] = 0;
  //printf(" %d; ", q[threadIdx.x]);

}

int main() {
	int *c;
	int *dev_c;
	c = (int*)malloc(N*sizeof(int));

	for (int i = 0; i < N; ++i)
		c[i] = rand() %10+1;

	for (int i = 0; i < N; ++i)
		printf(" %d; ", c[i]);

	hipMalloc((void**)&dev_c, N*sizeof(int));

	hipMemcpy(dev_c, c, N*sizeof(int), hipMemcpyHostToDevice);

	//foo<<<1, N>>>(dev_c);
	ESBMC_verify_kernel(foo,1,N,dev_c);

	hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

	printf ("\n");

	for (int i = 0; i < N; ++i){
		printf(" %d; ", c[i]);
		assert(c[i]!=0);
	}

	free(c);
	hipFree(dev_c);

	   return 0;
}
