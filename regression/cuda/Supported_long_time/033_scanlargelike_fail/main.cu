#include "hip/hip_runtime.h"
//fail
//--blockDim=32 --gridDim=64 --no-inline
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <assert.h>

#define N 2//32

__device__ void f(float *odata, int* ai) {
    int thid = threadIdx.x;
    *ai = thid;
    odata[*ai] = 2*threadIdx.x;
}

__global__ void k(float *g_odata) {
    int ai;
    f(g_odata,&ai);
}


int main(){
	float *d;
	float *dev_d;

	d = (float*)malloc(N*sizeof(float));
	hipMalloc ((void**) &dev_d, N*sizeof(float));

	hipMemcpy(dev_d, d, N*sizeof(float),hipMemcpyHostToDevice);

	//k <<<1,N>>>(dev_d);
	ESBMC_verify_kernel(k,1,N,dev_d);

	hipMemcpy(d,dev_d,N*sizeof(float),hipMemcpyDeviceToHost);

	printf("D: ");
	for (int i = 0; i < N; ++i) {
		printf(" %f		", d[i]);
		assert(!(d[i] == 2*i));
	}
	hipFree(dev_d);

	return 0;
}
