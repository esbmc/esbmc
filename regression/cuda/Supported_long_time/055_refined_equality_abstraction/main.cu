#include "hip/hip_runtime.h"
//pass
//--warp-sync=32 --blockDim=32 --gridDim=1 --equality-abstraction --no-inline
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime_api.h>

#define N 2//32

__global__ void foo(int * A, int * B) {
    A[threadIdx.x] = 1;
    volatile int x = A[threadIdx.x];
    B[threadIdx.x] = 1;
    volatile int y = A[threadIdx.x];
    assert(x==y);
}

int main(){
	int *a, *b;
	int *dev_a, *dev_b;
	int size = N*sizeof(int);

	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);

	a = (int*)malloc(size);
	b = (int*)malloc(size);

	for (int i = 0; i < N; i++)
		a[i] = 0;

	for (int i = 0; i < N; i++)
		b[i] = 2;

	hipMemcpy(dev_a,a,size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,size, hipMemcpyHostToDevice);

	//foo<<<1,N>>>(dev_a, dev_b);
		ESBMC_verify_kernel(foo, 1, N, dev_a, dev_b);

	hipMemcpy(a,dev_a,size,hipMemcpyDeviceToHost);
	hipMemcpy(b,dev_b,size,hipMemcpyDeviceToHost);

	free(a); free(b);

	hipFree(dev_a); hipFree(dev_b);

	return 0;
}
