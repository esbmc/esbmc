#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime_api.h"
#include <assert.h>

#define N 2//64

__global__ void foo(float* A) {

  if(blockIdx.x == 0)
	  A[threadIdx.x] = 50.f;
}

int main() {
	float *a = 0;
	float *dev_a;

	a = (float*)malloc(N*sizeof(float));

	hipMalloc((void**)&dev_a, N*sizeof(float));

	hipMemcpy(dev_a, a, N*sizeof(float), hipMemcpyHostToDevice);

		//foo<<<1, N>>>(dev_a);
		ESBMC_verify_kernel_f(foo, 1, N, dev_a);

	hipMemcpy(a, dev_a, N*sizeof(float), hipMemcpyDeviceToHost);

	printf ("%.1f", *a);

		assert(*a == 50.0);

	free(a);
	hipFree(dev_a);

	return 0;
}
