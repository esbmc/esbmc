#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>

#define N 2

__global__ void MoreSums(int *a, int *b, int *c){
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

int main(void){

	int *dev_a, *dev_b, *dev_c;
	int size = N*sizeof(int);

	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**) &dev_b, size);
	hipMalloc((void**)&dev_c,size);

	int a[N] = {1, 2};//, 1, 2, 3, 4};
	int b[N] = {1, 2};//, 1, 2, 3, 4};
	int c[N] = {1, 2};//, 1, 2, 3, 4};

	hipMemcpy(dev_a,&a,size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b,&b,size, hipMemcpyHostToDevice);

	//MoreSums<<<N,1>>>(dev_a,dev_b,dev_c);	//MODIFICAR: usar N threads em vez de blocos
	ESBMC_verify_kernel(MoreSums,N,1,dev_a,dev_b,dev_c);

	hipMemcpy(&c,dev_c,size,hipMemcpyDeviceToHost);

	//printf("\nResultado da soma de a e b eh:\n   ");

	for (int i = 0; i < N; i++){
		assert(c[i]==a[i]+b[i]);
	}

	hipFree(dev_a);
	hipFree(dev_c);
	hipFree(dev_b);

	return 0;
}
