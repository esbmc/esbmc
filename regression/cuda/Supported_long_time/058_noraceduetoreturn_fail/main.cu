#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime_api.h"
#include <assert.h>

#define N 2//64

__device__ int bar(float* A) {

  if(threadIdx.x != 0) {
	return 0;
  }

  return 1;

}

__global__ void foo(float* A) {

  int y = bar(A);
  A[threadIdx.x]=y;

}

int main(void){
	int i;
	float *A;
	float *dev_A;
	float size= N*sizeof(float);
	A=(float*)malloc(size);

	for(i=0;i<N;i++)
		A[i]=2;
			
	hipMalloc((void**)&dev_A,size);

	hipMemcpy(dev_A, A,size, hipMemcpyHostToDevice);

	//foo<<<1,N>>>(dev_A);
		ESBMC_verify_kernel_f(foo,1,N,dev_A);

	hipMemcpy(A,dev_A,size,hipMemcpyDeviceToHost);

	for(i=0;i<N;i++){
//		printf("%.2f ",A[i]);
		if (i!=0)
			assert(A[i]!=0);
		
	}
	hipFree(dev_A);
	free(A);
	return 0;
}
