#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>

#define N 2//64

__device__ int* bar(int* p) {

	//__ensures(__implies(__enabled(), __return_val_ptr() == p));
	return p;
}

__global__ void foo(int* p) {

  //bar(p)[threadIdx.x] = 0;
  *(bar(p)+threadIdx.x) = 2;
  //printf(" %d; ", bar(p)[threadIdx.x]);

}

int main() {
	int *c;
	int *dev_c;
	c = (int*)malloc(N*sizeof(int));

	for (int i = 0; i < N; ++i)
		c[i] = 1;

	hipMalloc((void**)&dev_c, N*sizeof(int));

	hipMemcpy(dev_c, c, N*sizeof(int), hipMemcpyHostToDevice);

	//foo<<<1, N>>>(dev_c);
	ESBMC_verify_kernel(foo,1,N,dev_c);

	hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

	printf ("\n");

	for (int i = 0; i < N; ++i){
		printf(" %d; ", c[i]);
		assert(c[i]==2);
	}

	free(c);
	hipFree(dev_c);

	   return 0;
}
