#include "hip/hip_runtime.h"
//xfail:BOOGIE_ERROR
//--warp-sync=32 --blockDim=32 --gridDim=1 --equality-abstraction --no-inline
//kernel.cu:10

#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <assert.h>
#define N 2//32

__global__ void foo(int * A) {
    A[0] = 1;
    A[1] = 1;
    A[threadIdx.x] = 0;
//__assert(A[0] == 1 | A[1] == 1 | A[2] == 1);
}

int main(){

	int *b;
	int *dev_b;

	b = (int*)malloc(N*sizeof(int));

	for (int i = 0; i < N; ++i){
		b[i] = 2;
	}

	hipMalloc((void**)&dev_b, N*sizeof(int));
	hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

	//foo<<<1,N>>>(dev_b);
	ESBMC_verify_kernel(foo, 1, N, dev_b);

	hipMemcpy(b, dev_b, N*sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < N; ++i){
		assert(b[i] == 0 || b[i] == 1);
	}

	free(b);
	hipFree(dev_b);
}
