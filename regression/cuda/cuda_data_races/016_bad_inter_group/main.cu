#include "hip/hip_runtime.h"
//xfail:BOOGIE_ERROR
//--blockDim=128 --gridDim=128 --warp-sync=32 --no-inline
//kernel.cu: error: possible read-write race on A
//It fail to dim >= 128, because it can't synchronize.
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#define N dim*dim
#define dim 2//128 //64

__global__ void foo(int* A) {

   A[ blockIdx.x*blockDim.x + threadIdx.x ] += (A[ (blockIdx.x + 1)*blockDim.x + threadIdx.x ]);

}

int main(){
 
    int *a;
    int *dev_a;
    int size = N*sizeof(int);

    hipMalloc((void**)&dev_a, size);

    a = (int*)malloc(N*size);

    for (int i = 0; i < N; i++)
        a[i] = i;

    hipMemcpy(dev_a,a,size, hipMemcpyHostToDevice);

    //foo<<<dim,dim>>>(dev_a);
	ESBMC_verify_kernel(foo, 1, dim, dev_a);

    hipMemcpy(a,dev_a,size,hipMemcpyDeviceToHost);

    free(a);
    hipFree(dev_a);

    return 0;
}
