#include "hip/hip_runtime.h"
// data-racer
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>

#define N 2

__global__ void foo(int	*p, int *ptr_a) {

	ptr_a = p + threadIdx.x;

}

int main() {
	int *c;
	int *dev_c;
	int *a;
	int *dev_a;


	c = (int*)malloc(N*sizeof(int));
	a = (int*)malloc(N*sizeof(int));

	for (int i = 0; i < N; ++i)
		c[i] = 2;

	hipMalloc((void**)&dev_c, N*sizeof(int));
	hipMalloc((void**)&dev_a, N*sizeof(int));

	hipMemcpy(dev_c, c, N*sizeof(int), hipMemcpyHostToDevice);

    //foo<<<1, N>>>(dev_c, dev_a);
	ESBMC_verify_kernel(foo,1,N,dev_c, dev_a);

	hipMemcpy(a, dev_a, N*sizeof(int), hipMemcpyDeviceToHost);

	free(a);
	free(c);
	hipFree(dev_a);
	hipFree(dev_c);

	return 0;
}
