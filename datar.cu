#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#define tid threadIdx.x
#define N 2

__device__ inline void inlined(int *A, int offset) {
int temp = A[tid + offset];
A[tid] += temp;
}

__global__ void inline_test(int *A, int offset) {
inlined(A, offset);
}

int main( ) {
int *a;
int *dev_a;
int size = N*sizeof(int);

hipMalloc((void**)&dev_a, size);

a = (int*)malloc(N*size);

for (int i = 0; i < N; i++)
a[i] = i;

hipMemcpy(dev_a,a,size, hipMemcpyHostToDevice);

ESBMC_verify_kernel_intt(inline_test, 1, N, dev_a, 2);

hipMemcpy(a,dev_a,size,hipMemcpyDeviceToHost);
free(a);
hipFree(dev_a);
return 0;
}
